#include "hip/hip_runtime.h"
#include "distconv/tensor/tensor.hpp"
#include "distconv/tensor/tensor_mpi.hpp"
#include "distconv/tensor/tensor_cuda.hpp"
#include "distconv/tensor/tensor_mpi_cuda.hpp"
#include "test_tensor.hpp"
#include "distconv/util/util_cuda.hpp"
#include "distconv/util/util_mpi.hpp"

#include <assert.h>
#include <iostream>
#include <vector>

using namespace distconv;
using namespace distconv::tensor;
using namespace distconv::util;

template <>
inline LocaleMPI get_locale() {
  LocaleMPI loc(MPI_COMM_WORLD);
  return loc;
}

__global__ void init_tensor(int *buf, Array<3> local_shape, Array<3> head_halo,
                            Array<3> tail_halo, index_t pitch, Array<3> global_shape,
                            Array<3> global_index_base) {
  Array<3> local_real_shape = local_shape + head_halo + tail_halo;
  for (index_t k = blockIdx.x; k < local_shape[2]; k += gridDim.x) {
    for (index_t j = 0; j < local_shape[1]; ++j) {
      for (index_t i = threadIdx.x; i < local_shape[0]; i += blockDim.x) {
        Array<3> local_idx = {i, j, k};
        size_t local_offset =
            get_offset(local_idx + head_halo, local_real_shape, pitch); // TODO: check
        Array<3> global_idx = global_index_base + local_idx;
        size_t global_offset = get_offset(
            global_idx, global_shape);
        buf[local_offset] = global_offset;
      }
    }
  }
}

__global__ void check_tensor(const int *buf, Array<3> local_shape,
                             Array<3> head_halo, Array<3> tail_halo, index_t pitch,
                             Array<3> global_shape, Array<3> global_index_base,
                             int *error_counter) {
  Array<3> local_real_shape = local_shape + head_halo + tail_halo;
  for (index_t k = blockIdx.x; k < local_shape[2]; k += gridDim.x) {
    for (index_t j = 0; j < local_shape[1]; ++j) {
      for (index_t i = threadIdx.x; i < local_shape[0]; i += blockDim.x) {
        Array<3> local_idx = {i, j, k};
        size_t local_offset =
            get_offset(local_idx + head_halo, local_real_shape, pitch); // todo: check
        Array<3> global_idx = global_index_base + local_idx;
        int global_offset = get_offset(global_idx, global_shape);
        int stored = buf[local_offset];
        if (stored != global_offset) {
          atomicAdd(error_counter, 1);
          printf("Error at (%lu, %lu, %lu)@(%lu, %lu, %lu); ref: %d, stored: %d\n",
                 global_idx[0], global_idx[1], global_idx[2],
                 i, j, k, global_offset, stored);
        }
      }
    }
  }
}

template <typename TensorType>
inline int test_data_access_mpi_cuda(const Shape &shape,
                                     const Distribution &dist) {
  using LocaleType = typename TensorType::locale_type;
  LocaleType loc = get_locale<LocaleType>();
  TensorType t = get_tensor<TensorType>(shape, loc, dist);
  MPIRootPrintStreamDebug() << "Shape: " << t.get_shape();
  MPIRootPrintStreamDebug() << "Distribution: " << t.get_distribution();
  MPIPrintStreamDebug() << "Local real shape: " << t.get_local_real_shape();

  assert0(t.allocate());

  //Array<3> local_shape = t.get_local_shape();
  index_t base_offset = t.get_local_offset();
  int *buf = t.get_buffer();
  assert_always(buf != nullptr);
  size_t pitch = t.get_pitch();
  util::MPIPrintStreamDebug() << "Base offset: " << base_offset
                              << ", global offset: " << t.get_global_index()
                              << ", pitch: " << pitch;

  init_tensor<<<4, 4>>>(buf, t.get_local_shape(), dist.get_head_overlap(),
                        dist.get_tail_overlap(), t.get_pitch(), t.get_shape(),
                        t.get_global_index());
  hipDeviceSynchronize();

  int error_counter = 0;
  int *error_counter_d;
  hipMalloc(&error_counter_d, sizeof(int));
  hipMemcpy(error_counter_d, &error_counter, sizeof(int),
             hipMemcpyDefault);
  check_tensor<<<1, 1>>>(buf, t.get_local_shape(), dist.get_head_overlap(),
                         dist.get_tail_overlap(), t.get_pitch(), t.get_shape(),
                         t.get_global_index(), error_counter_d);

  hipMemcpy(&error_counter, error_counter_d, sizeof(int),
             hipMemcpyDefault);
  assert0(error_counter);

  return 0;
}

template <typename TensorType>
int test_view_raw_ptr(const Shape &shape,
                      const Distribution &dist) {
  auto loc = get_locale<typename TensorType::locale_type>();
  auto t = get_tensor<TensorType>(shape, loc, dist);
  assert0(t.allocate());

  index_t base_offset = t.get_local_offset();
  int *buf = t.get_buffer();
  assert_always(buf);
  init_tensor<<<4, 4>>>(buf, t.get_local_shape(), dist.get_head_overlap(),
                        dist.get_tail_overlap(), t.get_pitch(), t.get_shape(),
                        t.get_global_index());
  hipDeviceSynchronize();
  using ConstTensorType = Tensor<typename TensorType::data_type,
                                 typename TensorType::locale_type,
                                 typename TensorType::allocator_type>;
  auto const_tensor_view = get_tensor<ConstTensorType>(shape, loc, dist);
  View(const_tensor_view, (const int*)buf);
  assert_always(const_tensor_view.get_const_buffer() == buf);
  int error_counter = 0;
  int *error_counter_d;
  hipMalloc(&error_counter_d, sizeof(int));
  hipMemcpy(error_counter_d, &error_counter, sizeof(int),
             hipMemcpyDefault);
  check_tensor<<<1, 1>>>(
      const_tensor_view.get_const_buffer(), const_tensor_view.get_local_shape(),
      dist.get_head_overlap(), dist.get_tail_overlap(), const_tensor_view.get_pitch(),
      const_tensor_view.get_shape(), const_tensor_view.get_global_index(),
      error_counter_d);
  hipMemcpy(&error_counter, error_counter_d, sizeof(int),
             hipMemcpyDefault);
  assert0(error_counter);
  return 0;
}

template <int ND, typename DataType>
__global__ void check_clear_halo(const DataType *buf, Array<ND> local_shape,
                                 int dim, int head_halo, int tail_halo,
                                 DataType default_value, int *error_counter) {
  const int tid = threadIdx.x;
  const int num_threads = blockDim.x;
  Array<ND> idx;
  idx[1] = blockIdx.x;
  idx[2] = blockIdx.y;
  if (ND == 4) {
    idx[3] = blockIdx.z;
  }

  for (int x = tid; x < local_shape[0]; x += num_threads) {
    idx[0] = x;
    int offset = get_offset(idx, local_shape);
    DataType v = buf[offset];
    if (idx[dim] < head_halo || idx[dim] >= local_shape[dim] - head_halo) { // TODO: check
      if (v != 0) {
        atomicAdd(error_counter, 1);
      }
    } else {
      if (v != default_value) {
        atomicAdd(error_counter, 1);
      }
    }
  }
}

template <int ND, typename TensorType>
int test_clear_halo(const Shape &shape,
                    const Distribution &dist) {
  const int num_dims = shape.num_dims();
  using DataType = typename TensorType::data_type;
  using LocaleType = typename TensorType::locale_type;
  LocaleType loc = get_locale<LocaleType>();
  TensorType t = get_tensor<TensorType>(shape, loc, dist);
  const auto local_real_shape = t.get_local_real_shape();
  util::MPIPrintStreamDebug() << "Shape: " << t.get_shape()
                              << ", local real shape: " << local_real_shape
                              << ", distribution: " << t.get_distribution();

  int error_counter = 0;
  int *error_counter_d;
  hipMalloc(&error_counter_d, sizeof(int));
  hipMemcpy(error_counter_d, &error_counter, sizeof(int),
             hipMemcpyDefault);

  assert0(t.allocate());
  auto *buf = t.get_buffer();
  DataType *h = new DataType[t.get_local_real_size()];
  DataType default_value = 1;
  for (size_t i = 0; i < t.get_local_real_size(); ++i) {
    h[i] = default_value;
  }
  for (int i = 0; i < num_dims; ++i) {
    hipMemcpy(buf, h, t.get_local_real_size() * sizeof(DataType),
               hipMemcpyDefault);
    t.clear_halo(i);
    dim3 gsize(local_real_shape[1], local_real_shape[2]);
    if (num_dims == 4) {
      gsize.z = local_real_shape[3];
    }
    check_clear_halo<ND, DataType>
        <<<gsize, 128>>>(buf, local_real_shape, i, dist.get_head_overlap(i),
                         dist.get_tail_overlap(i), 1, error_counter_d);
    hipMemcpy(&error_counter, error_counter_d, sizeof(int), hipMemcpyDefault);
    if (error_counter != 0) {
      util::MPIPrintStreamError() << error_counter << " errors at dimension ";
      DISTCONV_CHECK_CUDA(hipMemcpy(h, buf, sizeof(DataType) *
                                     t.get_local_real_size(), hipMemcpyDefault));
      std::ofstream out;
      std::stringstream file_path;
      file_path << "clear_halo_test_" << loc.get_rank();
      out.open(file_path.str(), std::ios::out | std::ios::trunc);
      for (size_t i = 0; i < t.get_local_real_size(); ++i) {
        out << h[i] << "\n";
      }
      out.close();
      return -1;
    }
#if 0
    DataType *result_h = new DataType[t.get_local_real_size()];
    DISTCONV_CHECK_CUDA(hipMemcpy(result_h, buf, sizeof(DataType) *
                                   t.get_local_real_size(), hipMemcpyDefault));
    std::ofstream out;
    std::stringstream file_path;
    file_path << "clear_halo_test_" << i << "_" << loc.get_rank();
    out.open(file_path.str(), std::ios::out | std::ios::trunc);
    for (size_t i = 0; i < t.get_local_real_size(); ++i) {
      out << result_h[i] << "\n";
    }
    out.close();
#endif
  }

  return 0;
}

/*
  Usage: mpirun -np N ./test_tensor_mpi_cuda, where N must be >= 8 and
  divisible by 8.
 */
int main(int argc, char *argv[]) {
  int dev = util::choose_gpu();
  hipSetDevice(dev);
  MPI_Init(&argc, &argv);
  int pid;
  int np;
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
  MPI_Comm_size(MPI_COMM_WORLD, &np);

  MPIPrintStreamInfo() << "Using device " << dev;

  constexpr int ND = 3;
  using DataType = int;

  using TensorMPI = Tensor<DataType, LocaleMPI, CUDAAllocator>;
  auto dist3 = Distribution::make_overlapped_distribution({2, 2, np / 4},
                                                          {1, 1, 0}, {1, 1, 0});
  auto dist4 = Distribution::make_overlapped_distribution(
      {2, 2, 2, np / 8}, {1, 1, 0, 0}, {1, 1, 0, 0});
  assert_always((np % 8) == 0 && (np >= 8));
  //Distribution<3> dist({1, 1, np}, {1, 1, 0});

  assert0(test_alloc<TensorMPI>(Shape({2, 2, 2}), dist3));
  MPIRootPrintStreamInfo() << "test_alloc success";

  assert0(test_data_access_mpi_cuda<TensorMPI>(Shape({2, 2, 2}), dist3));
  MPIRootPrintStreamInfo() << "test_data_access_mpi_cuda success";

  // Doesn't work with Spectrum-MPI
#if 0
  assert0(test_data_access_mpi_cuda<Tensor<DataType, LocaleMPI,
          CUDAPitchedAllocator>>(Shape({32, 32, 4}), dist3));
  MPIRootPrintStreamInfo() << "test_data_access_mpi_cuda with pitched memory success\n";
#endif

  assert0(test_view_raw_ptr<TensorMPI>(Shape({32, 32, 4}), dist3));

  MPIRootPrintStreamInfo() << "test_view_raw_ptr success";

  assert0(test_clear_halo<ND, TensorMPI>(Shape({32, 31, 4}), dist3));
  MPIRootPrintStreamInfo() << "test_clear_halo success";

  assert0(test_clear_halo<4, TensorMPI>(Shape({32, 31, 4, 8}), dist4));
  MPIRootPrintStreamInfo() << "test_clear_halo with 4D tensor success";

  MPI_Finalize();

  hipDeviceReset();
  return 0;
}
